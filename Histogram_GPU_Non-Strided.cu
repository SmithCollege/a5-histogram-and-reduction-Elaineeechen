
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define SIZE 10
#define RANGE 10
#define BLOCK_SIZE 1024

double get_clock() {
  struct timeval tv; int ok;
  ok = gettimeofday(&tv, (void *) 0);
  if (ok<0) { printf("gettimeofday error"); }
  return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}


__global__ void histogram(int* X, int* histo) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < SIZE) {
     atomicAdd(&(histo[X[i]]), 1);
  }
}


int main() {
  int *X, *histo;
  hipMallocManaged(&X, sizeof(int) * SIZE);
  for (int i = 0; i < SIZE; i++) {
    X[i] = rand() % RANGE;
  }
  hipMallocManaged(&histo, sizeof(int) * RANGE);
  for (int i = 0; i < RANGE; i++) {
    histo[i] = 0;
  }

  double t0 = get_clock();
  histogram<<<(SIZE + BLOCK_SIZE - 1)/BLOCK_SIZE, BLOCK_SIZE>>>(X, histo);
  hipDeviceSynchronize();
  double t1 = get_clock();

  for (int i = 0; i < RANGE; i++) {
    printf("Number %d: %d\n", i, histo[i]);
  }
  printf("Time: %f ns\n", 1000000000.0*(t1 - t0));
  printf("%s\n", hipGetErrorString(hipGetLastError()));

  return 0;
}
